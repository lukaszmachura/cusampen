
#include <hip/hip_runtime.h>
#include <math.h>
#include <stdio.h>
#include <getopt.h>

#define FILE_SIZE 666

#define gpuErrchk(ans) { gpuAssert((ans), __FILE__, __LINE__); }
inline void gpuAssert(hipError_t code, const char *file, int line, bool abort=true)
{
  if (code != hipSuccess){
    fprintf(stderr, "GPUassert: %s %s %d\n", hipGetErrorString(code), file, line);
    if (abort) exit(code);
  }
}


#define BLOCKSIZE 256

__device__ int d_m;
__device__ float d_r;

// Kernel functions
__global__ 
void reduce(int *g_idata, int *g_out) {
  __shared__ int sdata[BLOCKSIZE];
  // each thread loads one element from global to shared mem
  int i = blockIdx.x*blockDim.x + threadIdx.x;

  sdata[threadIdx.x] = g_idata[i];

  __syncthreads();
  // do reduction in shared mem
  for (int s=1; s < blockDim.x; s *=2)
  {
    int index = 2 * s * threadIdx.x;
    if (index < blockDim.x)
    {
      sdata[index] += sdata[index + s];
    }
    __syncthreads();
  }

  // write result for this block to global mem
  if (threadIdx.x == 0){
      atomicAdd(g_out, sdata[0]);
  }
}

__device__
bool is_equal(float a, float b, float eps)
{
  return fabs(a - b) < eps ? true : false;
}

__global__
void findvec(float *base_vec, float *in, int *mvec, int *mplus1vec)
{
  int index = blockIdx.x * blockDim.x + threadIdx.x;
  
  // check if vec of size m is similar
  int m = true;
  for (int i = 0; i < d_m; i++){
    if (!is_equal(in[index + i], base_vec[i], d_r)){
      m = false;
      break;
    }
  }
  if (m) mvec[index] = 1;

  // check if vec of size m + 1 is similar
  int mplus1 = true;
  for (int i = 0; i <= d_m; i++){ 
    if (!is_equal(in[index + i], base_vec[i], d_r)){
      mplus1 = false;
      break;
    }
  }
  if (mplus1) mplus1vec[index] = 1;
}

int load_data(char *fname, float *x)
{
  FILE *f = fopen(fname, "r");
  float buf;
  int i = 0;
  while(fscanf(f, "%f", &buf) > 0)
    x[i++] = buf;
  fclose(f);
  return i - 1;
}

int countlines(char *fname)
{
    FILE *f = fopen(fname, "r");
    if (f == NULL)
	return -1;

    char z;
    int linenumbers = 0;
    while((z = fgetc(f)) != EOF)
	if (z == '\n')
	    linenumbers++;
    fclose(f);
    return linenumbers;
}

// parameters
typedef struct {
  float r;                  // max distance
  int m;                    // embed dim
  char infile[FILE_SIZE];   //
  char outfile[FILE_SIZE];  //
} params;

static struct option options[] = {
    {"in", required_argument, NULL, 'i'},
    {"out", required_argument, NULL, 'o'},
    {"embed", required_argument, NULL, 'm'},
    {"radius", required_argument, NULL, 'r'},
};

void dump_params(params * p)
{
  fprintf(stdout,"#\n");
  fprintf(stdout,"#m:%d\n",p->m);
  fprintf(stdout,"#r:%f\n",p->r);
  fprintf(stdout,"#infile:%s\n",p->infile);
  fprintf(stdout,"#outfile:%s\n",p->outfile);
  fflush(stdout);
}

void usage(char **argv)
{
    printf("Usage: %s <params> \n\n", argv[0]);
    printf("Model params:\n");
    printf("   -m, --embed=INT        set the embedding dimension 'dim' to INT\n");
    printf("   -r, --radius=FLOAT     set the maximal distance between vectors\n");
    printf("                          to 'radius' to FLOAT\n");
    printf("   -i, --in=FILE_NAME     set the input data to FILE_NAME\n");
    printf("   -o, --out=FILE_NAME    set the output data to FILE_NAME\n");
    printf("\n");
}

void parse_arguments(int argc, char **argv, params *p)
{
  int c;

  while( (c = getopt_long(argc, argv, "m:r:i:o", options, NULL)) != EOF) {
    switch (c) {
      case 'm':
        sscanf(optarg, "%d", &(p->m));
        gpuErrchk(hipMemcpyToSymbol(HIP_SYMBOL(d_m), &(p->m), sizeof(int), 0, hipMemcpyHostToDevice));
        break;
      case 'r':
        sscanf(optarg, "%f", &(p->r));
        break;
      case 'i':
        strcpy(p->infile, optarg);
        break;
      case 'd':
        strcpy(p->outfile, optarg);
        break;
    }
  }
}


float sandard_deviation(float *x, int N)
{
  int i;
  float sd = 0, mean = 0;
  
  for (i = 0; i < N; ++i){
    mean += x[i];
    sd += x[i] * x[i];
  }
  
  mean /= N;
  sd = sd / N - mean * mean;
  
  return sqrt(sd);
}

int main(int argc, char **argv)
{
  params p = {
    0.2f,
    2,
    "data.dat",
    "out.dat"
  };
  
  parse_arguments(argc, argv, &p);
  dump_params(&p);
  
  int i;
  float *x;
  int *mvec, *mplus1vec;
  int *mmatches, *mplus1matches;
  float *base_vec, r, sd;
  int m;

  // data
  int N = countlines(p.infile);
  if (N <= 0) return N;

  // parrallelism
  int blockSize = BLOCKSIZE; //4 ;//256;
  int numBlocks = (N + blockSize - 1) / blockSize;

  // Allocate Unified Memory – accessible from CPU or GPU
  gpuErrchk(hipMallocManaged(&x, N * sizeof(float)));
  gpuErrchk(hipMallocManaged(&mmatches, sizeof(int)));
  gpuErrchk(hipMallocManaged(&mplus1matches, sizeof(int)));
  gpuErrchk(hipMallocManaged(&mvec, N * sizeof(int)));
  gpuErrchk(hipMallocManaged(&mplus1vec, N * sizeof(int)));

  // initialize data
  load_data(p.infile, x);
  sd = sandard_deviation(x, N);

  // Sampen algorithm initialisation
  m = p.m;
  r = p.r * sd;
  gpuErrchk(hipMemcpyToSymbol(HIP_SYMBOL(d_r), &r, sizeof(float), 0, hipMemcpyHostToDevice));

  // space in shared mem for base vec (m + 1)
  gpuErrchk(hipMallocManaged(&base_vec, (m + 1) * sizeof(float)));
  
  // search for EACH possible base vec
  int n_m = 0,
      n_mplus1 = 0;
  for (int ibv = 0; ibv < N - m - 1; ibv++){
      // clean storage
      for (i = 0; i < N; i++){ 
          mvec[i] = 0;
          mplus1vec[i] = 0;
      }
 
      // build temporary base vec of (m + 1) length
      for (i = ibv; i <= m; i++) base_vec[i] = x[i];

      // Run kernel on the GPU
      // find matches for temporary vec
      findvec<<<numBlocks, blockSize>>>(base_vec, x, mvec, mplus1vec);
      gpuErrchk(hipDeviceSynchronize());

      //reduce
      mmatches[0] = 0;
      reduce<<<numBlocks, blockSize>>>(mvec, mmatches);
      gpuErrchk(hipDeviceSynchronize());
      n_m += mmatches[0] - 1;

      mplus1matches[0] = 0;
      reduce<<<numBlocks, blockSize>>>(mplus1vec, mplus1matches);
      gpuErrchk(hipDeviceSynchronize());
      n_mplus1 += mplus1matches[0] - 1;
  }//end of search

  fprintf(stdout, "m vector matches: %d\n", n_m);
  fprintf(stdout, "(m+1) vector matches: %d\n", n_mplus1);
  fprintf(stdout, "ratio = n_{m+1}/n_m: %f\n", (float)n_mplus1/n_m);
  fprintf(stdout, "SampEn = -ln(ratio): %f\n", -log((float)n_mplus1/n_m));

  // Free memory
  gpuErrchk(hipFree(x));
  gpuErrchk(hipFree(mmatches));
  gpuErrchk(hipFree(mplus1matches));
  gpuErrchk(hipFree(mvec));
  gpuErrchk(hipFree(mplus1vec));
  
  return 0;
}


//not used kernel functions
//to be removed
__global__
void czek(int n, int *idx, int *str, int *bI, int *bD, int *tI, int *gD)
{
  int index = blockIdx.x * blockDim.x + threadIdx.x;
  int stride = blockDim.x * gridDim.x;
  for (int i = index; i < n; i += stride){
      idx[i] = index;
      str[i] = stride;
      bI[i] = blockIdx.x;
      bD[i] = blockDim.x;
      tI[i] = threadIdx.x;
      gD[i] = gridDim.x;
  }
}


